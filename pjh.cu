#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>

#include <float.h>

#include "kmacros.h"
#include "cuda-macros.h"

extern "C" {

__device__ double cuda_lchoose(double *cu_lft, int n, int k) {

  if (k < 0) return -DBL_MAX;
  if (n == 0 && k == 0) return 0;
  if (k > n || n == 0) return -DBL_MAX;

  return cu_lft[n] - (cu_lft[k] + cu_lft[n-k]);
  //return lgamma((double) n+1) - (lgamma((double) k+1) + lgamma((double) (n-k+1)));
}

  static double __attribute__((unused))lchoose(double *lft, int n, int k) {
  if (k < 0) return -DBL_MAX;
  if (n == 0 && k == 0) return 0;
  if (k > n || n == 0) return -DBL_MAX;

  return lft[n] - (lft[k] + lft[n-k]);
}

__global__ void cuda_pjh(double *pjh, double *fsp, double *cu_lft, int sample_size) {
  int i, j, h;
  j = blockDim.x * blockIdx.x + threadIdx.x;
  h = blockDim.y * blockIdx.y + threadIdx.y;

  if (j > sample_size) return;
  if (h > sample_size) return;

  for(i=j; i <= sample_size; i++) {
    pjh[j*(sample_size + 1) + h] += fsp[i]*exp(cuda_lchoose(cu_lft, i,j) + 
					       cuda_lchoose(cu_lft, sample_size - i, h - j) -
					       cuda_lchoose(cu_lft, sample_size, h));
  }
}

__global__ void cuda_pbk(double *pbk, double *pjh, int sample_size) {
  int b, k, q;

  b = blockDim.x * blockIdx.x + threadIdx.x;
  k = blockDim.y * blockIdx.y + threadIdx.y;
  
  if (b > sample_size) return;
  if (k >= sample_size) return;

  q = b - (sample_size - k) + 1;
  if (q > 0) {
    pbk[b*(sample_size+1) + k] += pjh[q*(sample_size+1) + k+1]*(q/(k+1.));
  }
  if (b < k+1) {
    pbk[b*(sample_size+1) + k] += pjh[b*(sample_size+1) + k+1]*((k+1.-b)/(k+1.));
  }
}

void cuda_calculate_pbk(double *pjh, double *pbk, double **r_cu_pbk, double **r_cu_fsp,
			double *fsp, double *cu_lft, int sample_size) {
  int n = sample_size;
  double *cu_pjh, *cu_pbk;
  double *cu_fsp;
  

  CUDA_MA(cu_fsp, sizeof(double)*(n+1));
  CUDA_MEMCPY_TO(cu_fsp, fsp, sizeof(double)*(n+1));

  memset(pjh, 0x0, sizeof(double)*(n+1)*(n+1));
  CUDA_MA(cu_pjh, sizeof(double)*(n+1)*(n+1));
  CUDA_MEMCPY_TO(cu_pjh, pjh, sizeof(double)*(n+1)*(n+1));

  memset(pbk, 0x0, sizeof(double)*(n+1)*(n+1));
  CUDA_MA(cu_pbk, sizeof(double)*(n+1)*(n+1));
  CUDA_MEMCPY_TO(cu_pbk, pbk, sizeof(double)*(n+1)*(n+1));

  dim3 dimBlock(8,8);
  dim3 dimGrid((n+1)/dimBlock.x + 1, (n+1)/dimBlock.y + 1);
  cuda_pjh<<<dimGrid, dimBlock>>>(cu_pjh, cu_fsp, cu_lft, sample_size);
  CUDA_MEMCPY_FROM(pjh, cu_pjh, sizeof(double)*(n+1)*(n+1));

  {
    dim3 dimBlock(8,8);
    dim3 dimGrid((n+1)/dimBlock.x + 1, (n+1)/dimBlock.y + 1);
    cuda_pbk<<<dimGrid, dimBlock>>>(cu_pbk, cu_pjh, sample_size);
  }
  CUDA_MEMCPY_FROM(pbk, cu_pbk, sizeof(double)*(n+1)*(n+1));

  CUDA_FREE(cu_pjh);

  if (r_cu_pbk != NULL)
    *r_cu_pbk = cu_pbk;
  else
    CUDA_FREE(cu_pbk);
  if (r_cu_fsp != NULL)
    *r_cu_fsp = cu_fsp;
  else
    CUDA_FREE(cu_fsp);
  
}

__device__ double p_kescape_gpu(double *cu_lft, int k, int n, double ad) {
  if (k == 0) return exp(-n*ad);
  return exp(cuda_lchoose(cu_lft,n,k) + k*log(1.0 - exp(-ad)) - (n-k)*ad);
}

static double __attribute__((unused))p_kescape_cpu(double *lft, int k, int n, double ad) {
  if (k == 0) return exp(-n*ad);
  return exp(lchoose(lft,n,k) + k*log(1.0 - exp(-ad)) - (n-k)*ad);
}

__global__ void cuda_spf(double *cu_spf, double *cu_pbk, double *cu_fsp, double *cu_x,
			 double *cu_lft, int spline_pts, int sample_size) {
  int s, f, k;
  double ad;
  
  s = blockDim.x * blockIdx.x + threadIdx.x;
  f = blockDim.y * blockIdx.y + threadIdx.y;

  if (s > spline_pts || f > sample_size) return;
  
  ad = exp(cu_x[s]);
  
  cu_spf[s*(sample_size+1) + f] = p_kescape_gpu(cu_lft, sample_size, sample_size, ad)*cu_fsp[f];
  for(k=0; k < sample_size; k++)
    cu_spf[s*(sample_size+1) + f] +=
      p_kescape_gpu(cu_lft, k, sample_size, ad)*cu_pbk[f*(sample_size+1) + k];
  
}

static double *mk_log_factorial_table(double **r_lft, int n) {
  double *t, *cu_lft;
  int i;
  
  MA(t, n+1, double);

  t[0] = 0;
  t[1] = 0;
  for(i=2;i <= n; i++)
    t[i] = lgamma(i+1.);
  //    t[i] = t[i-1] + log(i);

  CUDA_MA(cu_lft, sizeof(double)*(n+1));
  CUDA_MEMCPY_TO(cu_lft, t, sizeof(double)*(n+1));
  if (r_lft != NULL) *r_lft = t;
  else free(t);
  return cu_lft;
}

double *cuda_pjh_init_tables(int n) {
  return mk_log_factorial_table(NULL, n);
}

void cpu_calculate_pbk(double **pjh, double **pbk, double *fsp, double *lft, int sample_size) {
  int i, j, h, b, k, q;
  
  for(j=0;j<=sample_size;j++) {      
    for(h=0;h<=sample_size;h++) {
      pjh[j][h] = 0.;
      
      for(i=j;i<=sample_size;i++) {
	pjh[j][h] += fsp[i]*exp(lchoose(lft, i,j) + 
				lchoose(lft, sample_size - i, h - j) -
				lchoose(lft, sample_size, h));
      }
    }
  }

  for(b=0;b<=sample_size;b++) {
    for(k=0;k<sample_size;k++) {
      q = b - (sample_size - k) + 1;
      if (q > 0) {
	pbk[b][k] += pjh[q][k+1]*(q/(double) (k+1));
      }
      if (b < k+1) {
	pbk[b][k] += pjh[b][k+1]*((k+1-b)/(double) (k+1));
      }
    }
  }

}

void cuda_calculate_spline_pts(double *x, double *spf, double *cu_pbk,
			       double *cu_fsp, double *cu_lft,
			       int spline_pts, int sample_size) {
  double *cu_x, *cu_spf;
  
  memset(spf, 0x0, sizeof(double)*(spline_pts+1)*(sample_size+1));

  CUDA_MA(cu_x,   sizeof(double)*(spline_pts+1));
  CUDA_MA(cu_spf, sizeof(double)*(spline_pts+1)*(sample_size+1));

  CUDA_MEMCPY_TO(cu_x, x, sizeof(double)*(spline_pts+1));
  CUDA_MEMCPY_TO(cu_spf, spf, sizeof(double)*(spline_pts+1)*(sample_size+1));

  dim3 dimBlock(8,8);
  dim3 dimGrid((spline_pts + 1)/dimBlock.x + 1, (sample_size+1)/dimBlock.y + 1);
  cuda_spf<<<dimGrid, dimBlock>>>(cu_spf, cu_pbk, cu_fsp, cu_x, cu_lft, spline_pts,
				  sample_size);

  CUDA_MEMCPY_FROM(spf, cu_spf,  sizeof(double)*(spline_pts+1)*(sample_size+1));

  CUDA_FREE(cu_spf);
  CUDA_FREE(cu_x);
  CUDA_FREE(cu_pbk);
  CUDA_FREE(cu_fsp);

}
			       
void cpu_calculate_spline_pts(double *x, double **spf, double **pbk, double *fsp, double *lft,
			      int spline_pts, int sample_size) {
  int i, f, k;
  
  for(i=0; i<=spline_pts; i++) {
    double ad;

    ad = exp(x[i]);
    for(f=0;f<=sample_size;f++) {
      spf[i][f] = p_kescape_cpu(lft, sample_size, sample_size, ad)*fsp[f];
      //      fprintf(stderr,"%d\t%d\t%g\t%g\n",i,f,x[i],spf[i][f]);
      for(k=0;k<sample_size;k++) {
	double tmp;
	tmp = p_kescape_cpu(lft, k, sample_size, ad)*pbk[f][k];
	//	fprintf(stderr,"IFK: %d\t%d\t%g\t%g\t%g\n",i,f,x[i],pbk[f][k],tmp);
	spf[i][f] += tmp;
	//fprintf(stderr,"IFK: %d\t%d\t%g\t%g\t%g\n",i,f,x[i],pbk[f][k],spf[i][f]);
      }
      //fprintf(stderr,"%d\t%d\t%g\t%g\n",i,f,x[i],spf[i][f]);
    }
  }
}

#ifdef __cplusplus
}
#endif
#ifdef UNIT_TEST
#define LOG_AD_MIN (-20.0)
#define LOG_AD_MAX (4.0)
static double *neutral_fsp(int n) {
  int i;
  double *fsp, s;
  
  MA(fsp, (n+1), double);
  
  fsp[0] = 0.;
  fsp[n] = 0.;
  s = 0.;
  for(i=1;i<n;i++) {
    fsp[i] = 1./i;
    s += fsp[i];
  }
  for(i=1;i<n;i++) {
    fsp[i] /= s;
  }

  return fsp;
}

int main(int argc, char *argv[]) {
  double **pjh_gpu, **pjh_cpu, **pbk_gpu, **pbk_cpu, **spf_gpu, **spf_cpu;
  double *cu_lft, *cu_pbk, *cu_fsp, *lft, *fsp, *x;
  double cpu_ms, gpu_ms;
  int sample_size, i, j, h, b, k, spline_pts;
  struct timeval stopwatch_gpu, stopwatch_cpu;
  
  if (argc < 2) {
    fprintf(stderr,"\nSpecify sample size as first command line argument.\n");
    exit(-1);
  }
  sample_size = atoi(argv[1]);

  fsp = neutral_fsp(sample_size+1);
  cu_lft = mk_log_factorial_table(&lft, sample_size+1);

  MA(pjh_gpu, sample_size+1, double *);
  CA(pjh_gpu[0], (sample_size+1)*(sample_size+1), double);
  MA(pjh_cpu, sample_size+1, double *);
  CA(pjh_cpu[0], (sample_size+1)*(sample_size+1), double);

  MA(pbk_gpu, sample_size+1, double *);
  CA(pbk_gpu[0], (sample_size+1)*(sample_size+1), double);
  MA(pbk_cpu, sample_size+1, double *);
  CA(pbk_cpu[0], (sample_size+1)*(sample_size+1), double);
  for(i=1; i<=sample_size; i++) {
    pjh_gpu[i] = pjh_gpu[i-1] + (sample_size+1);
    pbk_gpu[i] = pbk_gpu[i-1] + (sample_size+1);
    pjh_cpu[i] = pjh_cpu[i-1] + (sample_size+1);
    pbk_cpu[i] = pbk_cpu[i-1] + (sample_size+1);
  }
  
  fprintf(stderr,"Computing pbk[][] for sample size %d -   ", sample_size);

  fprintf(stderr,"GPU: ");
  gettimeofday(&stopwatch_gpu, NULL);
  cuda_calculate_pbk(pjh_gpu[0], pbk_gpu[0], &cu_pbk, &cu_fsp, fsp, cu_lft, sample_size);
  gpu_ms = elapsed_time_ms(&stopwatch_gpu);
  fprintf(stderr," %1.1f ms", gpu_ms);

  fprintf(stderr,"\tCPU: ");
  gettimeofday(&stopwatch_cpu, NULL);
  cpu_calculate_pbk(pjh_cpu, pbk_cpu, fsp, lft, sample_size);
  cpu_ms = elapsed_time_ms(&stopwatch_cpu);
  fprintf(stderr," %1.1f ms\n", cpu_ms);

  double pjh_error = 0.;
  for(j=0; j<=sample_size; j++) {
    for(h=0; h<=sample_size; h++) {
      pjh_error += fabs(pjh_cpu[j][h] - pjh_gpu[j][h])/(pjh_cpu[j][h] + DBL_EPSILON);
      /*      fprintf(stdout,"%d\t%d\t%1.5f\t%1.5f\t%1.5f\t%1.5f\n", j, h, fsp[h], pjh_cpu[j][h], pjh_gpu[j][h],
      	      (pjh_cpu[j][h] - pjh_gpu[j][h])/(pjh_cpu[j][h]+DBL_EPSILON));*/
    }
  }
  pjh_error /= (sample_size+1)*(sample_size+1);

  double pbk_error = 0.;
  for(b=0; b <= sample_size; b++) {
    for(k=0; k <= sample_size; k++) {
      pbk_error += fabs(pbk_cpu[b][k] - pbk_gpu[b][k])/(pbk_cpu[b][k] + DBL_EPSILON);
      /*      fprintf(stdout,"%d\t%d\t%1.5f\t%1.5f\t%1.5f\t%1.5f%%\n", b, k, fsp[k],
	      pbk_cpu[b][k], pbk_gpu[b][k],
	      (pbk_cpu[b][k] - pbk_gpu[b][k])/(pbk_cpu[b][k]+DBL_EPSILON)*100.);*/
    }
  }
  pbk_error /= (sample_size+1)*(sample_size+1);
  fprintf(stderr,"pjh[][] average %% deviance of GPU calculation from CPU calculation: %g\n",
	  pjh_error);
  fprintf(stderr,"pbk[][] average %% deviance of GPU calculation from CPU calculation: %g\n",
	  pbk_error);
  fprintf(stderr,"GPU is %1.1fX faster than CPU\n", cpu_ms/gpu_ms);

  spline_pts = 200;
  MA(x, spline_pts + 1, double);
  MA(spf_cpu, (spline_pts + 1), double *);
  CA(spf_cpu[0], (spline_pts+1)*(sample_size+1), double);
  MA(spf_gpu, (spline_pts+1), double *);
  CA(spf_gpu[0], (spline_pts+1)*(sample_size+1), double);
  for(i=1; i <= spline_pts; i++) {
    spf_cpu[i] = spf_cpu[i-1] + (sample_size+1);
    spf_gpu[i] = spf_gpu[i-1] + (sample_size+1);
  }
  for(i=0; i <= spline_pts; i++)
    x[i] = LOG_AD_MIN + i*(LOG_AD_MAX - LOG_AD_MIN)/(double) (spline_pts+1);

  fprintf(stderr,"Computing spline pts for sample size %d -   ", sample_size);

  fprintf(stderr,"GPU: ");
  gettimeofday(&stopwatch_gpu, NULL);
  cuda_calculate_spline_pts(x, spf_gpu[0], cu_pbk, cu_fsp, cu_lft, spline_pts,sample_size); 
  gpu_ms = elapsed_time_ms(&stopwatch_gpu);
  fprintf(stderr," %1.1f ms", gpu_ms);

  fprintf(stderr,"\tCPU: ");
  gettimeofday(&stopwatch_cpu, NULL);
  cpu_calculate_spline_pts(x, spf_cpu, pbk_cpu, fsp, lft, spline_pts, sample_size);
  cpu_ms = elapsed_time_ms(&stopwatch_cpu);
  fprintf(stderr," %1.1f ms\n", cpu_ms);

  double spf_error = 0.;
  for(int f=0; f< sample_size; f++) {
    for(i=0; i <= spline_pts; i++) {
      //      fprintf(stderr,"%d\t%d\t%1.3f\t%g\t%g\n", f, i, x[i], spf_cpu[i][f], spf_gpu[i][f]);
      spf_error += fabs(spf_cpu[i][f] - spf_gpu[i][f])/(spf_cpu[i][f] + DBL_EPSILON);
    }
  }
  spf_error /= (spline_pts+1)*(sample_size+1);
  fprintf(stderr,"spf[][] average deviance of GPU calculation from CPU calculation: %g\n",
	  spf_error);
  fprintf(stderr,"GPU is %1.1fX faster than CPU\n", cpu_ms/gpu_ms);
  
  free(pjh_gpu[0]);
  free(pjh_gpu);

  free(pjh_cpu[0]);
  free(pjh_cpu);

  free(pbk_gpu[0]);
  free(pbk_gpu);

  free(pbk_cpu[0]);
  free(pbk_cpu);

  CUDA_FREE(cu_lft);
  free(lft);
  free(fsp);
  return 0;
}
#endif

