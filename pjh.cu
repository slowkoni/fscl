#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>

#include <float.h>

#include "kmacros.h"
#include "cuda-macros.h"

__device__ double cuda_lchoose(double *cu_lft, int n, int k) {

  if (k < 0) return -DBL_MAX;
  if (n == 0 && k == 0) return 0;
  if (k > n || n == 0) return -DBL_MAX;

  return cu_lft[n] - (cu_lft[k] + cu_lft[n-k]);
  //return lgamma((double) n+1) - (lgamma((double) k+1) + lgamma((double) (n-k+1)));
}

double lchoose(double *lft, int n, int k) {
  if (k < 0) return -DBL_MAX;
  if (n == 0 && k == 0) return 0;
  if (k > n || n == 0) return -DBL_MAX;

  return lft[n] - (lft[k] + lft[n-k]);
}

__global__ void cuda_pjh(double *pjh, double *fsp, double *cu_lft, int sample_size) {
  int i, j, h;
  j = blockDim.x * blockIdx.x + threadIdx.x;
  h = blockDim.y * blockIdx.y + threadIdx.y;

  if (j > sample_size) return;
  if (h > sample_size) return;

  for(i=j; i <= sample_size; i++) {
    pjh[j*(sample_size + 1) + h] += fsp[i]*exp(cuda_lchoose(cu_lft, i,j) + 
					       cuda_lchoose(cu_lft, sample_size - i, h - j) -
					       cuda_lchoose(cu_lft, sample_size, h));
  }
}

__global__ void cuda_pbk(double *pbk, double *pjh, int sample_size) {
  int b, k, q;

  b = blockDim.x * blockIdx.x + threadIdx.x;
  k = blockDim.y * blockIdx.y + threadIdx.y;
  
  if (b > sample_size) return;
  if (k >= sample_size) return;

  q = b - (sample_size - k) + 1;
  if (q > 0) {
    pbk[b*(sample_size+1) + k] += pjh[q*(sample_size+1) + k+1]*(q/(k+1.));
  }
  if (b < k+1) {
    pbk[b*(sample_size+1) + k] += pjh[b*(sample_size+1) + k+1]*((k+1.-b)/(k+1.));
  }
}
   
void cuda_calculate_pjh(double *pjh, double *fsp, double *cu_lft, int sample_size) {
  int n = sample_size;
  double *cu_pjh, *cu_pbk, *pbk;
  double *cu_fsp;
  

  CUDA_MA(cu_fsp, sizeof(double)*(n+1));
  CUDA_MEMCPY_TO(cu_fsp, fsp, sizeof(double)*(n+1));

  memset(pjh, 0x0, sizeof(double)*(n+1)*(n+1));
  CUDA_MA(cu_pjh, sizeof(double)*(n+1)*(n+1));
  CUDA_MEMCPY_TO(cu_pjh, pjh, sizeof(double)*(n+1)*(n+1));

  MA(pbk, (n+1)*(n+1), double);
  memset(pbk, 0x0, sizeof(double)*(n+1)*(n+1));
  CUDA_MA(cu_pbk, sizeof(double)*(n+1)*(n+1));
  CUDA_MEMCPY_TO(cu_pbk, pjh, sizeof(double)*(n+1)*(n+1));

  dim3 dimBlock(8,8);
  dim3 dimGrid((n+1)/dimBlock.x + 1, (n+1)/dimBlock.y + 1);
  cuda_pjh<<<dimGrid, dimBlock>>>(cu_pjh, cu_fsp, cu_lft, sample_size);

#if 1
  cuda_pbk<<<dimGrid, dimBlock>>>(cu_pbk, cu_pjh, sample_size);
#endif

  CUDA_MEMCPY_FROM(pjh, cu_pjh, sizeof(double)*(n+1)*(n+1));
  CUDA_MEMCPY_FROM(pbk, cu_pbk, sizeof(double)*(n+1)*(n+1));
  
  CUDA_FREE(cu_pbk);
  CUDA_FREE(cu_pjh);
  CUDA_FREE(cu_fsp);
}

static double *mk_log_factorial_table(double **r_lft, int n) {
  double *t, *cu_lft;
  int i;
  
  MA(t, n+1, double);

  t[0] = 0;
  t[1] = 0;
  for(i=2;i <= n; i++)
    t[i] = lgamma(i+1.);
  //    t[i] = t[i-1] + log(i);

  CUDA_MA(cu_lft, sizeof(double)*(n+1));
  CUDA_MEMCPY_TO(cu_lft, t, sizeof(double)*(n+1));
  if (r_lft != NULL) *r_lft = t;
  else free(t);
  return cu_lft;
}

double *cuda_pjh_init_tables(int n) {
  return mk_log_factorial_table(NULL, n);
}

#ifdef UNIT_TEST
static double *neutral_fsp(int n) {
  int i;
  double *fsp, s;
  
  MA(fsp, (n+1), double);
  
  fsp[0] = 0.;
  fsp[n] = 0.;
  s = 0.;
  for(i=1;i<n;i++) {
    fsp[i] = 1./i;
    s += fsp[i];
  }
  for(i=1;i<n;i++) {
    fsp[i] /= s;
  }

  return fsp;
}

void cpu_calculate_pjh(double **pjh, double *fsp, double *lft, int sample_size) {
  int i, j, h;
  
  for(j=0;j<=sample_size;j++) {      
    for(h=0;h<=sample_size;h++) {
      pjh[j][h] = 0.;
      
      for(i=j;i<=sample_size;i++) {
	pjh[j][h] += fsp[i]*exp(lchoose(lft, i,j) + 
				lchoose(lft, sample_size - i, h - j) -
				lchoose(lft, sample_size, h));
      }
    }
  }

}

int main(int argc, char *argv[]) {
  double **pjh_gpu, **pjh_cpu;
  double *cu_lft, *lft, *fsp;
  double cpu_ms, gpu_ms;
  int sample_size, i, j, h;
  struct timeval stopwatch_gpu, stopwatch_cpu;
  
  if (argc < 2) {
    fprintf(stderr,"\nSpecify sample size as first command line argument.\n");
    exit(-1);
  }
  sample_size = atoi(argv[1]);

  fsp = neutral_fsp(sample_size+1);
  cu_lft = mk_log_factorial_table(&lft, sample_size+1);

  MA(pjh_gpu, sample_size+1, double *);
  CA(pjh_gpu[0], (sample_size+1)*(sample_size+1), double);
  MA(pjh_cpu, sample_size+1, double *);
  CA(pjh_cpu[0], (sample_size+1)*(sample_size+1), double);
  for(i=1; i<=sample_size; i++) {
    pjh_gpu[i] = pjh_gpu[i-1] + (sample_size+1);
    pjh_cpu[i] = pjh_cpu[i-1] + (sample_size+1);
  }
  
  fprintf(stderr,"Computing pjh[][] for sample size %d -   ", sample_size);

  fprintf(stderr,"GPU: ");
  gettimeofday(&stopwatch_gpu, NULL);
  cuda_calculate_pjh(pjh_gpu[0], fsp, cu_lft, sample_size);
  gpu_ms = elapsed_time_ms(&stopwatch_gpu);
  fprintf(stderr," %1.1f ms", gpu_ms);

  fprintf(stderr,"\tCPU: ");
  gettimeofday(&stopwatch_cpu, NULL);
  cpu_calculate_pjh(pjh_cpu, fsp, lft, sample_size);
  cpu_ms = elapsed_time_ms(&stopwatch_cpu);
  fprintf(stderr," %1.1f ms\n", cpu_ms);

  double pjh_error = 0.;
  for(j=0; j<=sample_size; j++) {
    for(h=0; h<=sample_size; h++) {
      pjh_error += fabs(pjh_cpu[j][h] - pjh_gpu[j][h])/(pjh_cpu[j][h] + DBL_EPSILON);
      //      fprintf(stdout,"%d\t%d\t%1.5f\t%1.5f\t%1.5f\t%1.5f\n", j, h, fsp[h], pjh_cpu[j][h], pjh_gpu[j][h],
      //	      (pjh_cpu[j][h] - pjh_gpu[j][h])/(pjh_cpu[j][h]+DBL_EPSILON));
    }
  }
  pjh_error /= (sample_size+1)*(sample_size+1);
  fprintf(stderr,"Average %% deviance of GPU calculation from CPU calculation: %g\n",
	  pjh_error*100.);
  fprintf(stderr,"GPU is %1.1fX faster than CPU\n", cpu_ms/gpu_ms);
  
  free(pjh_gpu[0]);
  free(pjh_gpu);

  free(pjh_cpu[0]);
  free(pjh_cpu);

  CUDA_FREE(cu_lft);
  free(lft);
  free(fsp);
  return 0;
}
#endif
